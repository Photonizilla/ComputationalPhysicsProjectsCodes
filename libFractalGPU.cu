#include "hip/hip_runtime.h"
// Compile to library:
// nvcc -Xcompiler -fPIC -c libFractalGPU.cu -o libFractalGPU.o -lm
// nvcc -shared -Xcompiler -fPIC libFractalGPU.o -o libFractalGPU.so -lcudart
//

#include <math.h>
#include <hip/hip_complex.h>
#include "error_check.h"
#include <stdio.h>

__device__ static const unsigned char listR[] = {0, 137, 76, 137, 20, 66, 88, 116, 115, 168, 203, 255, 248, 240, 238, 235, 136};
__device__ static const unsigned char listG[] = {0, 43, 31, 61, 50, 148, 191, 249, 246, 247, 250, 251, 214, 153, 110, 64, 33};
__device__ static const unsigned char listB[] = {0, 142, 141, 246, 245, 247, 249, 253, 156, 77, 80, 84, 72, 55, 43, 37, 17};

__global__ static void rootAtPixel(unsigned char* rootResult, const int N, const double Lx, const double Rx, const double Ly, const double Ry) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (col >= N || row >= N)
        return;
    hipDoubleComplex Z = make_hipDoubleComplex((Rx - Lx) / (N - 1) * col + Lx, (Ry - Ly) / (N - 1) * row + Ly);
    hipDoubleComplex solution1 = make_hipDoubleComplex(1, 0);
    hipDoubleComplex solution2 = make_hipDoubleComplex(-0.5, sqrt(0.75));
    hipDoubleComplex solution3 = make_hipDoubleComplex(-0.5, -sqrt(0.75));
    const double r = (Rx - Lx < Ry - Ly) ? (Rx - Lx) : (Ry - Ly);
    while (hipCabs(hipCsub(Z, solution1)) > r / N && hipCabs(hipCsub(Z, solution2)) > r / N && hipCabs(hipCsub(Z, solution3)) > r / N) {
        Z = hipCsub(Z, hipCdiv(hipCsub(hipCmul(hipCmul(Z, Z), Z), make_hipDoubleComplex(1, 0)), hipCmul(hipCmul(make_hipDoubleComplex(3, 0), Z), Z)));
    }
    double imagPart = hipCimag(Z);
    if (imagPart < -0.5) {
        rootResult[(row * N + col) * 3 + 0] = 85;
        rootResult[(row * N + col) * 3 + 1] = 177;
        rootResult[(row * N + col) * 3 + 2] = 71;
    } else if (imagPart > 0.5) {
        rootResult[(row * N + col) * 3 + 0] = 63;
        rootResult[(row * N + col) * 3 + 1] = 132;
        rootResult[(row * N + col) * 3 + 2] = 247;
    } else {
        rootResult[(row * N + col) * 3 + 0] = 193;
        rootResult[(row * N + col) * 3 + 1] = 76;
        rootResult[(row * N + col) * 3 + 2] = 66;
    }
}

__global__ static void orderAtPixel(unsigned char* orderResult, const int N, const double Lx, const double Rx, const double Ly, const double Ry) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (col >= N || row >= N)
        return;
    hipDoubleComplex Z = make_hipDoubleComplex((Rx - Lx) / (N - 1) * col + Lx, (Ry - Ly) / (N - 1) * row + Ly);
    unsigned char count = 0;
    hipDoubleComplex solution1 = make_hipDoubleComplex(1, 0);
    hipDoubleComplex solution2 = make_hipDoubleComplex(-0.5, sqrt(0.75));
    hipDoubleComplex solution3 = make_hipDoubleComplex(-0.5, -sqrt(0.75));
    const double r = (Rx - Lx < Ry - Ly) ? (Rx - Lx) : (Ry - Ly);
    while (hipCabs(hipCsub(Z, solution1)) > r / N && hipCabs(hipCsub(Z, solution2)) > r / N && hipCabs(hipCsub(Z, solution3)) > r / N) {
        Z = hipCsub(Z, hipCdiv(hipCsub(hipCmul(hipCmul(Z, Z), Z), make_hipDoubleComplex(1, 0)), hipCmul(hipCmul(make_hipDoubleComplex(3, 0), Z), Z)));
        if (count < 255)
            count++;
    }
    if (count <= 16) {
        orderResult[(row * N + col) * 3 + 0] = listR[count];
        orderResult[(row * N + col) * 3 + 1] = listG[count];
        orderResult[(row * N + col) * 3 + 2] = listB[count];
    } else {
        orderResult[(row * N + col) * 3 + 0] = (unsigned char)(136.0 * exp(-0.0625 * (count - 16)));
        orderResult[(row * N + col) * 3 + 1] = (unsigned char)(33.0 * exp(-0.0625 * (count - 16)));
        orderResult[(row * N + col) * 3 + 2] = (unsigned char)(17.0 * exp(-0.0625 * (count - 16)));
    }
}

extern "C" {

void fractalRootCalc(unsigned char* rootOutput, const int N, const double Lx, const double Rx, const double Ly, const double Ry) {
    unsigned char* rootColorsDevice = NULL;
    hipMalloc((void**)&rootColorsDevice, N * N * 3 * sizeof(unsigned char));
    CHECK(hipGetLastError());
    
    dim3 dimGrid(ceil(N / 16.0), ceil(N / 16.0), 1);
    dim3 dimBlock(16, 16, 1);
    printf("GPU Started...\n");
    rootAtPixel<<<dimGrid, dimBlock>>>(rootColorsDevice, N, Lx, Rx, Ly, Ry);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    
    hipMemcpy(rootOutput, rootColorsDevice, N * N * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);
    CHECK(hipGetLastError());
    
    hipFree(rootColorsDevice);
    rootColorsDevice = NULL;
    CHECK(hipGetLastError());
    CHECK(hipDeviceReset());
    printf("fractalCalc Completed!\n");
}

void fractalOrderCalc(unsigned char* orderOutput, const int N, const double Lx, const double Rx, const double Ly, const double Ry) {
    unsigned char* orderColorsDevice = NULL;
    hipMalloc((void**)&orderColorsDevice, N * N * 3 * sizeof(unsigned char));
    CHECK(hipGetLastError());
    
    dim3 dimGrid(ceil(N / 16.0), ceil(N / 16.0), 1);
    dim3 dimBlock(16, 16, 1);
    printf("GPU Started...\n");
    orderAtPixel<<<dimGrid, dimBlock>>>(orderColorsDevice, N, Lx, Rx, Ly, Ry);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    
    hipMemcpy(orderOutput, orderColorsDevice, N * N * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);
    CHECK(hipGetLastError());
    
    hipFree(orderColorsDevice);
    orderColorsDevice = NULL;
    CHECK(hipGetLastError());
    CHECK(hipDeviceReset());
    printf("fractalCalc Completed!\n");
}

}

